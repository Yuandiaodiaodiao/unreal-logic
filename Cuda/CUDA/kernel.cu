#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include "hip/hip_runtime.h"
#include ""
#include<iostream>
#include <stdio.h>
#include<sstream>
#include<string>
#include <time.h>
using namespace std;
hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);

__global__ void addKernel(int* c, const int* a, const int* b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

__global__ void GateKernel(int* NewNodeStatus, const int* NodeStatus, const int* GateType)
{
	int i = threadIdx.x;
	if (GateType[i] == 0) {
		//or
		NewNodeStatus[i] = (NodeStatus[i] & 1) | (NodeStatus[i] & 2 >> 1);
	}
	else if (GateType[i] == 1) {
		//and
		NewNodeStatus[i] = (NodeStatus[i] & 1) & (NodeStatus[i] & 2 >> 1);
	}
	else if (GateType[i] == 2) {
		//not
		NewNodeStatus[i] = ~(NodeStatus[i] & 1);
	}
}

__global__ void AndGateKernel(int* NewNodeStatus, const int* NodeStatus)
{
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	NewNodeStatus[i] = (NodeStatus[i] & 1) & (NodeStatus[i] & 2 >> 1);
}
__global__ void OrGateKernel(int* NewNodeStatus, const int* NodeStatus)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	NewNodeStatus[i] = (NodeStatus[i] & 1) | (NodeStatus[i] & 2 >> 1);
}
__global__ void NotGateKernel(int* NewNodeStatus, const int* NodeStatus)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	NewNodeStatus[i] = ~(NodeStatus[i] & 1);
}

void initCuda() {
	int dev = 0;
	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, dev);


}
int baseblock = 1024;
hipError_t GateWithCuda(int* OldStatus, int* NewStatus,int arraySize,int GateType,int* dev_Old,int* dev_New) {
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (GateType == 0) {
		AndGateKernel <<< arraySize/ baseblock, baseblock >>> (dev_New, dev_Old);
	}
	else if (GateType == 1) {
		OrGateKernel <<< arraySize / baseblock, baseblock >>> (dev_New, dev_Old);
	}
	else if (GateType == 2) {
		NotGateKernel <<< arraySize / baseblock, baseblock >>> (dev_New, dev_Old);
	}
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		cout <<(int) dev_New << endl;
		cout <<(int)dev_Old << endl;
		fprintf(stderr, "GateWithCuda launch failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}

	return cudaStatus;

}
const char* doPreformanceTest(int* AndOldStatus, int* AndNewStatus,
	int* OrOldStatus, int* OrNewStatus,
	int* NotOldStatus, int* NotNewStatus, int arraySize, int testTimes) {
	int* Old[3] = { AndOldStatus,OrOldStatus,NotOldStatus };
	int* New[3] = { AndNewStatus,OrNewStatus,NotNewStatus };
	
	
	
	
	int** devon= new int* [6];
	for (int a = 0; a <= 2; ++a) {
		devon[a*2] = 0;
		devon[a*2+1] = 0;
		hipMalloc((void**)&devon[a * 2], arraySize * sizeof(int));
		hipMalloc((void**)&devon[a * 2 + 1], arraySize * sizeof(int));
		hipError_t cudaStatus;
		cudaStatus=hipMemcpy(devon[a * 2], Old[a], arraySize * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
		}
	}
	clock_t start, end;
	start = clock();

	for (int a = 0; a < testTimes; ++a) {
		//async kernel
		for (int gate = 0; gate <= 2; ++gate) {
			GateWithCuda(Old[gate], New[gate], arraySize, gate, devon[gate * 2], devon[gate * 2 + 1]);
		}
		//fprintf(stderr, "完成第%d轮 \n", a);
		for (int gate = 0; gate <= 2; ++gate) {
			hipMemcpy(New[gate], devon[gate *2+1], arraySize * sizeof(int), hipMemcpyDeviceToHost);
			//write back
			memcpy(Old[gate], New[gate], arraySize * sizeof(int));
			//swap old and new
			swap(devon[gate * 2], devon[gate * 2 + 1]);

		}
	}

	end = clock();
	printf("time=%f\n", (double)(end - start)*1.0 / CLK_TCK*1000);
	for (int a = 0; a <= 2; ++a) {
		hipFree(devon[a * 2]);
		hipFree(devon[a * 2 + 1]);
	}
	
	{
		hipError_t cudaStatus;
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed! %s", hipGetErrorString(cudaStatus));
		}
	}
	
	return "";
}

const char* doAdd() {
	const int arraySize = 5;
	const int a[arraySize] = { 1, 2, 3, 4, 5 };
	const int b[arraySize] = { 10, 20, 30, 40, 50 };
	int c[arraySize] = { 0 };
	string sout = "";
	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return sout.c_str();
	}
	char file_path_buffer[100];
	sprintf(file_path_buffer, "{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
		c[0], c[1], c[2], c[3], c[4]);
	sout += file_path_buffer;
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return sout.c_str();
	}
	char* temp = new char[sout.length() + 2];
	sprintf(temp, "%s", sout.c_str());
	return temp;
}
int main()
{


	initCuda();
	int** xx = new int* [6];
	int arraySize = 0;
	std::cout << "start!\n" << endl;
	cin >> arraySize;
	arraySize *= baseblock;
	
	for (int a = 0; a <= 5; ++a) {
		xx[a] = new int[arraySize];
		for (int z = 0; z < arraySize;++z) {
			xx[a][z] = rand();
		}
	}
	long long anss = 0;
	const char* ss = doPreformanceTest(xx[0], xx[1], xx[2], xx[3], xx[4], xx[5], arraySize, 100);
	for (int a = 1; a <= 5; a+=2) {
		for (int z = 0; z < arraySize; ++z) {
			anss += xx[a][z];
		}
	}
	
	cout << "ans is " << anss << " !" << endl;
	std::cout << "Hello World!\n" << ss;
	for (int a = 0; a <= 5; ++a) {
		delete xx[a];
	}


	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel <<<1, size >>> (dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
